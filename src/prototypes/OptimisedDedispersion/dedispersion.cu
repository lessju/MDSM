
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "time.h"
#include "string.h"

#ifndef DEDISP_THREADS
    #define DEDISP_THREADS  128  
#endif

#ifndef DEDISP_DMS
    #define DEDISP_DMS      32   
#endif

#define NUMREG 8
#define DIVINT 4
#define DIVINDM 32

float fch1 = 418, foff = -0.01953125, tsamp = 0.0000512, dmstep = 0.1, startdm = 0;
int nchans = 1024, nsamp = 131072, tdms = 4096, compare = 0;

// ======================== CUDA HELPER FUNCTIONS ==========================

// Error checking function
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) _cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    _cudaCheckError( __FILE__, __LINE__ )

inline void _cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void _cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }   
#endif

    return; 
}

// ======================= Wes' Dedispersion Loop =============================
__global__ void cache_dedispersion(float *output, float *input, float *dm_shifts, 
                                   const int nsamp, const int nchans, const float mstartdm, 
                                   const float mdmstep, const int maxshift)
{
	int   shift;	
	float local_kernel_t[NUMREG];

	int t  = blockIdx.x * NUMREG * blockDim.x  + threadIdx.x;
	
	// Initialise the time accumulators
	for(int i = 0; i < NUMREG; i++) local_kernel_t[i] = 0.0f;

	float shift_temp = mstartdm + ((blockIdx.y * blockDim.y + threadIdx.y) * mdmstep);
	
	// Loop over the frequency channels.
    for(int c = 0; c < nchans; c++) 
    {
		// Calculate the initial shift for this given frequency
		// channel (c) at the current despersion measure (dm) 
		// ** dm is constant for this thread!!**
		shift = (c * (nsamp + maxshift) + t) + (dm_shifts[c] * shift_temp);
		
        #pragma unroll
		for(int i = 0; i < NUMREG; i++) {
			local_kernel_t[i] += input[shift + (i * DIVINT) ];
		}
	}

	// Write the accumulators to the output array. 
    #pragma unroll
	for(int i = 0; i < NUMREG; i++) {
		output[((blockIdx.y * DIVINDM) + threadIdx.y)* nsamp + (i * DIVINT) + (NUMREG * DIVINT * blockIdx.x) + threadIdx.x] = local_kernel_t[i];
	}
}

// ======================= Optimised Dedispersion Loop 1 =======================
__global__ void dedisperse_loop1(const float* __restrict__ input, float* __restrict__ output, 
							     const int* __restrict__ all_delays, const unsigned nchans, 
                                 const unsigned nsamp, const int maxshift, const int tdms)
{
	// Shared memory buffer to store channel vector
	extern __shared__ float vector[];

	// Each thread will process a number of DM values associated with one time sample
	register float accumulators[DEDISP_DMS];

	// Initialise shared memory store for dispersion delays
	__shared__ int delays[DEDISP_DMS];

	// Initialise accumulators
	for(unsigned d = 0; d < DEDISP_DMS; d++) accumulators[d] = 0;

	// Loop over all frequency channels
	for(unsigned c = 0; c < nchans; c++)
	{
		// Synchronise threads before updating dispersion delays
		__syncthreads();

		// Load all the shifts associated with this threadblock DM-range for the current channel
		int inshift = all_delays[c * tdms + blockIdx.y * DEDISP_DMS];
		if (threadIdx.x < DEDISP_DMS)
			delays[threadIdx.x] = all_delays[c * tdms + blockIdx.y * DEDISP_DMS + threadIdx.x] - inshift;
		
		// Synchronise threads
		__syncthreads();

		// We'll need to load the channel vector (which will be larger than threadDim
		// due to dispersion
		for(unsigned s = threadIdx.x; 
					 s < blockDim.x + delays[DEDISP_DMS - 1]; 
					 s += blockDim.x)
			vector[s] = input[(maxshift + nsamp) * c + blockIdx.x * blockDim.x + inshift + s];

		// Synchronise threads
		__syncthreads();

		// Loop over DM values associated with current threadblock and update accumulators
		// Manual unlooping of four to overlap shared memory requests
		#pragma unroll
		for(int d = 0; d < DEDISP_DMS; d ++)
	            accumulators[d]  += vector[threadIdx.x + delays[d]];
	}

	// All done, store result to global memory
    #pragma unroll
	for(unsigned d = 0; d < DEDISP_DMS; d++)
		output[(blockIdx.y * DEDISP_DMS + d) * nsamp + blockIdx.x * blockDim.x + threadIdx.x] = accumulators[d];
}

// ======================= Main Program =======================

// Process command-line parameters
void process_arguments(int argc, char *argv[])
{
    int i = 1;
    
    while(i < argc) {
       if (!strcmp(argv[i], "-nchans"))
           nchans = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsamp"))
           nsamp = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-dmstep"))
           dmstep = atof(argv[++i]);
       else if (!strcmp(argv[i], "-startdm"))
           startdm = atof(argv[++i]);
       else if (!strcmp(argv[i], "-tdms"))
           tdms = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-tsamp"))
           tsamp = atof(argv[++i]);
       else if (!strcmp(argv[i], "-foff"))
           foff = -atof(argv[++i]);
       else if (!strcmp(argv[i], "-fch1"))
           fch1 = atof(argv[++i]);
       else if (!strcmp(argv[i], "-compare"))
           compare = 1;
       i++;
    }
}

// Fill buffer with data (blocking call)
void generate_data(float* buffer, int nsamp, int nchans)
{
    for(int i = 0; i < nsamp * nchans; i++)
        buffer[i] = 0.1;
}

// DM delay calculation
float dmdelay(float f1, float f2)
{
  return(4148.741601 * ((1.0 / f1 / f1) - (1.0 / f2 / f2)));
}

int main(int argc, char *argv[])
{
   float *input, *output, *d_input, *d_output, *d_delays;
   int maxshift;

   process_arguments(argc, argv);

    // Calculate temporary DM-shifts
    float *dmshifts = (float *) malloc(nchans * sizeof(float));
    for (unsigned i = 0; i < nchans; i++)
          dmshifts[i] = dmdelay(fch1 + (foff * i), fch1) / tsamp;

    // Calculate maxshift
    maxshift = ceil(dmshifts[nchans - 1] * (startdm + dmstep * tdms));

    // Allocate and initialise arrays
    input = (float *) malloc( (nsamp + maxshift) * nchans * sizeof(float));
    memset(input, 0, (nsamp + maxshift) * nchans * sizeof(float));
    output = (float *) malloc( nsamp * tdms * sizeof(float));

    // Initialise CUDA stuff
    CudaSafeCall(hipSetDevice(1));
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    hipEvent_t event_start, event_stop;
    float timestamp, kernelTime;

    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop);

    printf("dedisp_threads: %d, dedisp_dm: %d\n", DEDISP_THREADS, DEDISP_DMS);
    printf("nsamp: %d, nchans: %d, tsamp: %f, startdm: %f, dmstep: %f, tdms: %d, fch1: %f, foff: %f, maxshift: %d\n",
           nsamp, nchans, tsamp, startdm, dmstep, tdms, fch1, foff, maxshift);

    printf("Memory requirements: Input: %.2f MB, Output: %.2f MB \n", nchans * (nsamp + maxshift) * sizeof(float) / (1024.0 * 1024),
                                                                      tdms * nsamp * sizeof(float) / (1024.0 * 1024.0));

    // Allocate CUDA memory and copy dmshifts
    CudaSafeCall(hipMalloc((void **) &d_input, (nsamp + maxshift) * nchans * sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_output, nsamp * tdms * sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_delays, nchans * sizeof(float)));
    CudaSafeCall(hipMemset(d_output, 0, nsamp * tdms * sizeof(float)));
	memset(output, 0, nsamp * tdms * sizeof(float));

    time_t start = time(NULL);

    // Copy input to GPU
    hipEventRecord(event_start, 0);
    CudaSafeCall(hipMemcpy(d_input, input, (nsamp + maxshift) * nchans * sizeof(float), hipMemcpyHostToDevice) );    
    CudaSafeCall(hipMemcpy(d_delays, dmshifts, nchans * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied to GPU in: %lf\n", timestamp);

    // Dedisperse using Wes' kernel
    if (compare)
    {
        int num_reg         = NUMREG;
        int divisions_in_t  = DIVINT;
        int divisions_in_dm = DIVINDM;
        int num_blocks_t    = nsamp / (divisions_in_t * num_reg);
        int num_blocks_dm   = tdms / divisions_in_dm;

        dim3 threads_per_block(divisions_in_t, divisions_in_dm);
        dim3 num_blocks(num_blocks_t,num_blocks_dm); 

        hipEventRecord(event_start, 0);	
        cache_dedispersion<<< num_blocks, threads_per_block >>>
                          (d_output, d_input, d_delays, nsamp, nchans, startdm, dmstep, maxshift);
        hipEventRecord(event_stop, 0);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&timestamp, event_start, event_stop);
        printf("Performed Cache Dedispersion %lf\n", timestamp);
	
    }   
	// Cacluate the extra shared memory required to store shifts
	unsigned shift = round(dmshifts[nchans - 1] * (startdm + tdms * dmstep)) - 
                     round(dmshifts[nchans - 1] * (startdm + (tdms - DEDISP_DMS) * dmstep));

	// Pre-compute channel and DM specific shifts beforehand on CPU
	// This only needs to be computed once for the entire execution
	int *all_shifts = (int *) malloc(nchans * tdms * sizeof(int));
	for(unsigned c = 0; c < nchans; c++)
		for (unsigned d = 0; d < tdms; d++)
			all_shifts[c * tdms + d] = (int) (dmshifts[c] * (startdm + (d * dmstep)));

	int *d_all_shifts;
	CudaSafeCall(hipMalloc((void **) &d_all_shifts, nchans * tdms * sizeof(int)));
	CudaSafeCall(hipMemcpy(d_all_shifts, all_shifts, nchans * tdms * sizeof(int), hipMemcpyHostToDevice) );  

	dim3 gridDim(ceil(nsamp / (1.0 * DEDISP_THREADS)), ceil(tdms / (1.0 * DEDISP_DMS)));  
    hipEventRecord(event_start, 0);
    dedisperse_loop1 <<< gridDim, DEDISP_THREADS, (DEDISP_THREADS + shift) * sizeof(float) >>> 
       (d_input, d_output, d_all_shifts, nchans, nsamp, maxshift, tdms);

    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Perform Shared Memory Dedispersion [v1] in: %lf\n", timestamp);
    kernelTime = timestamp;

    // Copy output from GPU
    hipEventRecord(event_start, 0);
    CudaSafeCall(hipMemcpy(output, d_output, nsamp * tdms * sizeof(float), hipMemcpyDeviceToHost) );    
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied from GPU in: %lf\n", timestamp);

    printf("Performance: %lf Gflops\n", (nchans * tdms * 6) * (nsamp * 1.0 / kernelTime / 1.0e6));
}

