#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <sys/time.h>
#include "time.h"
#include <gsl/gsl_multifit.h>
#include <math.h>
#include "file_handler.h"
 

#define BANDPASS_THREADS 256

//char *filename = "/data/Data/SETI/B1839+56_8bit.fil";
//char *filename = "/data/Data/SETI/samplePulsar.fil";
char *filename = "/home/lessju/Kepler_Pulsar_RFI.dat";
int nchans = 2048, nsamp = 16384, ncoeffs = 12;
float channel_thresh = 2.5, spectrum_thresh = 0.5;
unsigned channel_block = 16;

// ======================== CUDA HELPER FUNCTIONS ==========================

// Error checking function
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) _cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    _cudaCheckError( __FILE__, __LINE__ )

inline void _cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void _cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

// ========================= CPU HELPER FUNCTIONS ===========================
void read_data(float *buffer, unsigned nsamp, unsigned nchans)
{
    // Read file
    float *tempBuff = (float *) malloc(nsamp * nchans * sizeof(float));
    FILE *fp = fopen(filename, "rb");
    
    // Read header
    read_header(fp);

    int num_read = read_block(fp, 32, tempBuff, nchans * nsamp);
    fclose(fp);

    // Transpose data
    unsigned i, j;
    for(i = 0; i < nchans; i++)    
        for(j = 0; j < nsamp; j++)
            buffer[i * nsamp + j] = tempBuff[j * nchans + i];

    free(tempBuff);

    if (num_read != nsamp * nchans)
    {
        printf("Seems there's not enough data in the file\n");
        exit(0);
    }
}


// ==========================================================================

// Compute power from input complex values
// A[N] = A[N].x * A[N].x + A[N].y * A[N].y
// Performed in place (data will still consume 32-bits in GPU memory)
__global__ void power(float *data, unsigned nsamp)
{
    for(unsigned s = blockIdx.x * blockDim.x + threadIdx.x; 
                 s < nsamp;
                 s += gridDim.x * blockDim.x)
    {
        short2 value = *((short2 *) &data[s]);
        data[s] = value.x * value.x + value.y * value.y;
    }
}

// Compute the first pass for bandpass generation
// Sum along the channels to get averaged sum, which will be use
// to compute the polynomial co-efficients and fit
__global__ void bandpass_power_sum(float *input, double *bandpass, unsigned nsamp)
{
    // Declare shared memory to store temporary mean and stddev
    __shared__ double local_sum[BANDPASS_THREADS];

    // Initialise shared memory
    local_sum[threadIdx.x] = 0;

    // Synchronise threads
    __syncthreads();

    // Loop over samples
    for(unsigned s = threadIdx.x;
                 s < nsamp; 
                 s += blockDim.x)

        local_sum[threadIdx.x] += input[blockIdx.x * nsamp + s]; 

    // Synchronise threads
    __syncthreads();

    // Use reduction to calculate block mean and stddev
	for (unsigned i = BANDPASS_THREADS / 2; i >= 1; i /= 2)
	{
		if (threadIdx.x < i)
            local_sum[threadIdx.x]  += local_sum[threadIdx.x + i];
		
		__syncthreads();
	}

    // Finally, return temporary sum
    if (threadIdx.x == 0)
        bandpass[blockIdx.x] = local_sum[0] / nsamp;
}

// --------------------- Perform rudimentary RFI clipping: clip channels ------------------------------
__global__ void channel_clipper(float *input, double *bandpass, float bandpass_mean, unsigned channel_block, 
                                unsigned nsamp, unsigned nchans, float channelThresh)
{
    __shared__ float local_mean[BANDPASS_THREADS];

    // 2D Grid, Y-dimension handles channels, X-dimension handles spectra
    float bp_value = __double2float_rz(bandpass[blockIdx.y]);
    local_mean[threadIdx.x] = 0;

    // Load all required value to shared memory
    for(unsigned s = threadIdx.x; 
                 s < channel_block; 
                 s += blockDim.x)
        local_mean[threadIdx.x] += input[blockIdx.y * nsamp + blockIdx.x * channel_block + s] - bp_value;

    __syncthreads();

    // Perform reduction-sum to calculate the mean for current channel block
    for(unsigned i = BANDPASS_THREADS / 2; i > 0; i /= 2)
    {
        if (threadIdx.x < i)
            local_mean[threadIdx.x] += local_mean[threadIdx.x + i];
        __syncthreads();
    }

    // Compute channel mean
    if (threadIdx.x == 0)
        local_mean[0] /= channel_block;

    __syncthreads();

    // This should be handled as a shared-memory broadcast, check
    // Check if channel block exceeds acceptable limit, if not flag
    if (local_mean[0] > channelThresh)
        for(unsigned s = threadIdx.x; 
                     s < channel_block; 
                     s += blockDim.x)
        input[blockIdx.y * nsamp + blockIdx.x * channel_block + s] -= (input[blockIdx.y * nsamp + blockIdx.x * channel_block + s] - bp_value);

}

// --------------------- Perform rudimentary RFI clipping: clip spectra ------------------------------
__global__ void spectrum_clipper(float *input, double *bandpass, double *corrected_bandpass, float bandpass_mean,
                                 unsigned nsamp, unsigned nchans, float spectrumThresh)
{
    // First pass done, on to second step
    // Second pass: Perform wide-band RFI clipping
    for(unsigned s = blockIdx.x * blockDim.x + threadIdx.x;
                 s < nsamp;
                 s += gridDim.x * blockDim.x)
    {
        // For each spectrum, we need to calculate the mean
        float spectrum_mean = 0;

        // All these memory accesses should be coalesced (as thread-spectrum mapping is contiguous)    
        for(unsigned c = 0; c < nchans; c++)
            spectrum_mean += (input[c * nsamp + s] - __double2float_rz(bandpass[c]));
        spectrum_mean /= nchans;

        // We have the spectrum mean, check if it satisfies spectrum threshold
        if (spectrum_mean > spectrumThresh)
            // Spectrum is RFI, clear (zero out for now)
            for(unsigned c = 0; c < nchans; c++)
                input[c * nsamp + s] = 500;//bandpass[c];

        // Check for singe high-intensity pixels
//        for(unsigned c = 0; c < nchans; c++)
//            if (input[c*nsamp+s] > 2*spectrumThresh)
//                input[c*nsamp+s] = 1e4;//bandpass[c];
    }
}

// ============================= CPU BANDPASS FIT ====================================
bool polynomialfit(int obs, int degree, double *dx, double *dy, double *store) /* n, p */
{
    gsl_multifit_linear_workspace *ws;
    gsl_matrix *cov, *X;
    gsl_vector *y, *c;
    double chisq;
 
    int i, j;
 
    X = gsl_matrix_alloc(obs, degree);
    y = gsl_vector_alloc(obs);
    c = gsl_vector_alloc(degree);
    cov = gsl_matrix_alloc(degree, degree);
 
    for(i=0; i < obs; i++) 
    {
        gsl_matrix_set(X, i, 0, 1.0);
        for(j=0; j < degree; j++)
            gsl_matrix_set(X, i, j, pow(dx[i], j));
        gsl_vector_set(y, i, dy[i]);
    }
 
    ws = gsl_multifit_linear_alloc(obs, degree);
    gsl_multifit_linear(X, y, c, cov, &chisq, ws);
 
    /* store result ... */
    for(i=0; i < degree; i++)
        store[i] = gsl_vector_get(c, i);
 
    gsl_multifit_linear_free(ws);
    gsl_matrix_free(X);
    gsl_matrix_free(cov);
    gsl_vector_free(y);
    gsl_vector_free(c);
    return true; // Check the result to know if the fit is good (conv matrix)
}

// Process command-line parameters
void process_arguments(int argc, char *argv[])
{
    int i = 1;
    
    while((fopen(argv[i], "r")) != NULL)
        i++;

    while(i < argc) {
       if (!strcmp(argv[i], "-nchans"))
           nchans = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsamp"))
           nsamp = atoi(argv[++i]);
       i++;
    }
}

// Main function
int main(int argc, char *argv[])
{
    unsigned i, j;

    struct timeval start, end;
    long mtime, seconds, useconds; 

	hipEvent_t event_start, event_stop;
	float timestamp;
	hipEventCreate(&event_start); 
	hipEventCreate(&event_stop); 

    // Allocate and initialise CPU and GPU memory for data and bandpass
    float *buffer; double *bandpass, *corrected_bandpass;
    CudaSafeCall(hipHostMalloc((void **) &buffer, nchans * nsamp * sizeof(float), hipHostMallocPortable));
    CudaSafeCall(hipHostMalloc((void **) &corrected_bandpass, nchans * sizeof(double), hipHostMallocPortable));
    CudaSafeCall(hipHostMalloc((void **) &bandpass, nchans * sizeof(double), hipHostMallocPortable));

    // Read data from file and reset initialise bandpass to 0
    read_data(buffer, nsamp, nchans);
    memset(bandpass, 0, nchans * sizeof(double));

    float *d_buffer; double *d_bandpass, *d_corrected_bandpass;
    hipMalloc((void **) &d_buffer, nchans * nsamp * sizeof(float));
	hipMalloc((void **) &d_corrected_bandpass, nchans * sizeof(double) );
	hipMalloc((void **) &d_bandpass, nchans * sizeof(double) );
    hipMemset(d_bandpass, 0, nchans * sizeof(double));

    // Copy input buffer to GPU memory
    hipEventRecord(event_start, 0);
    hipMemcpy(d_buffer, buffer, nchans * nsamp * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Copied data to GPU in: %lf\n", timestamp);

    // First pass for bandpass fitting, compute sums
    hipEventRecord(event_start, 0);
    bandpass_power_sum<<<nchans, BANDPASS_THREADS>>>(d_buffer, d_bandpass, nsamp);
    hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Computed bandpass sum in : %lf\n", timestamp);

    // Second pass, calculate bandpass co-efficients and bandpass fit
    gettimeofday(&start, NULL);

    CudaSafeCall(hipMemcpy(bandpass, d_bandpass, nchans * sizeof(double), hipMemcpyDeviceToHost));

    double X[nchans], coeffs[ncoeffs];
    for(i = 0; i < nchans; i++) X[i] = 0 + i / (1.0 * nchans);

    // Fit polynomial using GNU Scientific Library
    polynomialfit(nchans, ncoeffs, X, bandpass, coeffs);

    FILE *fpb = fopen("Test_bandpass_fit.dat", "wb");
    fwrite(bandpass, sizeof(double), nchans, fpb);

    // Generate 1D polynomial using bandpass co-efficients
    // We also need the fit-corrected bandpass to compute the bandpass RMS
    memcpy(corrected_bandpass, bandpass, nchans * sizeof(double)); // Copy bandpass to corrected_bandpass
    memset(bandpass, 0, nchans * sizeof(double));
    for(i = 0; i < nchans; i++)
    {
        for(j = 0; j < ncoeffs; j++)
            bandpass[i] += coeffs[j] * pow(X[i], j);
        corrected_bandpass[i] -= bandpass[i];
    }

    fwrite(bandpass, sizeof(double), nchans, fpb);
    fwrite(corrected_bandpass, sizeof(double), nchans, fpb);
    fclose(fpb);

    // Compute bandpass RMS
    float corr_bandpass_mean = 0, corr_bandpass_std = 0, corr_bandpass_rms = 0;
    float bandpass_mean = 0, bandpass_std = 0, bandpass_rms = 0;

    // First iteration to compute mean
    for(i = 0; i < nchans; i++)
    {
        bandpass_mean += bandpass[i];
        bandpass_rms += bandpass[i] * bandpass[i];
        corr_bandpass_mean += corrected_bandpass[i];
        corr_bandpass_rms += corrected_bandpass[i] * corrected_bandpass[i];
    }
    bandpass_mean /= nchans;
    bandpass_rms = sqrt(bandpass_rms / nchans);
    corr_bandpass_mean /= nchans;
    corr_bandpass_rms = sqrt(corr_bandpass_rms / nchans);
    
    // Second iteration, compute standard deviation
    for(i = 0; i < nchans; i++)
    {
        bandpass_std += (bandpass[i] - bandpass_mean) * (bandpass[i] - bandpass_mean);
        corr_bandpass_std += (bandpass[i] - corr_bandpass_mean) * (bandpass[i] - corr_bandpass_mean);
    }
    bandpass_std = sqrt(bandpass_std / nchans);
    corr_bandpass_std = sqrt(corr_bandpass_std / nchans);

    // Copy bandpass back to GPU memory
    CudaSafeCall(hipMemcpy(d_corrected_bandpass, corrected_bandpass, nchans * sizeof(double), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_bandpass, bandpass, nchans * sizeof(double), hipMemcpyHostToDevice));

    gettimeofday(&end, NULL);
    seconds  = end.tv_sec  - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;
    mtime = ((seconds) * 1000 + useconds/1000.0) + 0.5;
    printf("Calculated Bandpass co-efficients and fit in : %ldms\n", mtime);

    // RFI Clipping, launch GPU-RFI clipper
    channel_thresh  *= corr_bandpass_rms;
    spectrum_thresh *=  corr_bandpass_rms;

    printf("Corrected Bandpass RMS: %f\nBandpass RMS: %f\nBandpass Std: %f\nSpectrum Thresh: %f\nChannel Thresh: %f\n", corr_bandpass_rms, bandpass_rms, bandpass_std, spectrum_thresh, channel_thresh);

    hipEventRecord(event_start, 0);
    channel_clipper<<< dim3(nsamp/channel_block, nchans), BANDPASS_THREADS >>>
                   (d_buffer, d_bandpass, bandpass_mean, channel_block, nsamp, nchans, channel_thresh);

    hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Clipped channels in : %lf\n", timestamp);

    hipEventRecord(event_start, 0);
    spectrum_clipper<<< nsamp / BANDPASS_THREADS, BANDPASS_THREADS >>>
                   (d_buffer, d_bandpass, d_corrected_bandpass, bandpass_mean, nsamp, nchans, spectrum_thresh);

    hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Clipped spectra in : %lf\n", timestamp);

    // Copy result back to CPU memory
    hipEventRecord(event_start, 0);
    CudaSafeCall(hipMemcpy(buffer, d_buffer, nsamp * nchans * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(event_stop, 0);
	hipEventSynchronize(event_stop);
	hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Copied results back to CPU memory in : %lf\n", timestamp);

    // Write buffer to file
    FILE *fp = fopen("Test_bandpass.dat", "wb");
    fwrite(buffer, nchans * nsamp, sizeof(float), fp);
    fclose(fp);
    
}

    
