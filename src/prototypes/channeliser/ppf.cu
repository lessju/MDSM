#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "time.h"
#include "string.h"
#include "math.h"

#include "hipfft/hipfft.h"

#define NTAPS 32

unsigned nsubs = 32, nsamp = 65536, nbeams = 1;
unsigned nchans = 1024;

// ======================== CUDA HELPER FUNCTIONS ==========================

// Error checking function
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) _cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    _cudaCheckError( __FILE__, __LINE__ )

inline void _cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void _cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }   
#endif

    return; 
}

inline void Exit()
{
	exit(0);
}

// ======================= Channelisation Kernels =============================
// NOTE: For this kernel, nchans <= blockDim.x
__global__ void ppf_fir(hipComplex *input, const float *window, const unsigned nsamp, 
                        const unsigned nsubs, const unsigned nbeams, const unsigned nchans) 
{
    // Subband moves in y dimension
    // Beam moves in z dimension

    // Declare shared memory to store window coefficients
     extern __shared__ float coeffs[];

    // Each thread is associated with a particular channel and sample
    unsigned channel_num = threadIdx.x % nchans;
    unsigned sample_num = threadIdx.x / nchans;
    unsigned sample_shift = (blockDim.x / nchans) == 0 ? 1 : blockDim.x / nchans;

    // Loop over channels (in cases where nchans > blockDim.x)
    for(unsigned c = channel_num;
                 c < nchans;
                 c += blockDim.x)
    {
        // FIFO buffer is stored in local register array
        hipComplex fifo[NTAPS] = { 0 };  

        // Initialise FIFO with first NTAPS values
        unsigned index = blockIdx.y * nsubs * nsamp * nchans + blockIdx.x * nsamp * nchans;
        for(unsigned i = 0; i < NTAPS - 1; i++)
            fifo[i] = input[index + nchans * i + c];

        // Load window coefficients to be used by each thread
        for(unsigned i = 0; i < NTAPS; i++)
            coeffs[threadIdx.x + i * blockDim.x] = window[i * nchans + c];

        // Synchronise threads
        __syncthreads();

        // Loop over all samples for current channel
        // Start at the (NTAPS-1)th sample, in order to use FIFO buffer
		#pragma unroll 2
        for(unsigned s = sample_num + NTAPS - 1;
                     s < nsamp;
                     s += sample_shift)
        {
            // Declare output value
            hipComplex output = { 0, 0 };

            // Store new value in FIFO buffer
            fifo[NTAPS - 1] = input[index + s * nchans + c];

            // Apply window
			#pragma unroll NTAPS
            for (unsigned t = 0; t < NTAPS; t++)
            {
                float coeff = coeffs[threadIdx.x + blockDim.x * t];
                output.x += fifo[t].x * coeff;
                output.y += fifo[t].y * coeff;
            }

			// Store output to global memory
            input[index + s * nchans + c] = output;

            // Re-arrange FIFO buffer
			#pragma unroll NTAPS
			for(unsigned i = 0; i < NTAPS - 1; i++)
				fifo[i] = fifo[i + 1];
        } 
    }
}

__global__ void fix_channelisation(float2 *input, float *output, unsigned nsamp, unsigned nchans, unsigned nbeams, 
                                   unsigned subchans, unsigned start_chan)
{    
    // Time changes in the x direction
    // Channels change along the y direction. Indexing start at start_chan
    // Beams change along the z direction
    // Each thread processes one sample

	// Get index to start of current channelised block
    // ThreadIdx.x is the nth channel formed in this block
	unsigned long indexIn  = blockIdx.z * nchans * nsamp + (start_chan + blockIdx.y) * nsamp + threadIdx.x;
    unsigned long indexOut =  (blockIdx.y * subchans + threadIdx.x) * nbeams + blockIdx.z;

    for(unsigned s = blockIdx.x;
                 s < nsamp / subchans;
                 s += gridDim.x)
    {
        float2 value = input[indexIn + s * subchans];
        output[s * nbeams * gridDim.y * subchans + indexOut] = sqrtf(value.x * value.x + value.y * value.y);
    }
}

// ======================= Main Program =======================================

// Process command-line parameters
void process_arguments(int argc, char *argv[])
{
    int i = 1;
    
    while(i < argc) {
       if (!strcmp(argv[i], "-nchans"))
           nchans = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsamp"))
           nsamp = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nbeams"))
           nbeams = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsubs"))
           nsubs = atoi(argv[++i]);
       i++;
    }
}

// Notes: in real-time application we'll need to buffer the last ntaps * nchans values
// for use in the next buffer. Strategy:
// - Keep extra values in a separate buffer (multiple device memcopies)
// - Use this buffer to load the initial values, normal data is loaded in main loop
int main(int argc, char *argv[])
{
    hipComplex *input, *d_input;
	float *output, *d_output, *weights, *d_weights;

    process_arguments(argc, argv);

    printf("nsamp: %d, nsubs: %d, nbeams: %d, nchans: %d\n", nsamp, nsubs, nbeams, nchans);
    printf("Memory requirements: Input: %.2f MB, Output: %.2f \n", nsubs * nbeams * nsamp * sizeof(hipComplex) / (1024.0 * 1024), 0);

	// Set 8-byte shared memory
//	hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(ppf_fir), hipSharedMemBankSizeEightByte );

    // Allocate and initialise arrays
    input   = (hipComplex *) malloc(nsamp * nsubs * nbeams * sizeof(hipComplex));
	output  = (float *) malloc(nsamp * nsubs * nbeams * sizeof(float));
    weights = (float *) malloc(nchans * NTAPS * sizeof(float));
    memset(input,  0, nsamp * nsubs * nbeams * sizeof(hipComplex));
	memset(output, 0, nsamp * nsubs * nbeams * sizeof(float));
    memset(weights, 0, nchans * NTAPS * sizeof(float));

	// Load coefficients
    char filename[256];
    sprintf(filename, "coeff_%d_%d.dat", NTAPS, nchans);
    FILE *fp = fopen(filename, "rb");
    fread(weights, sizeof(float), NTAPS * nchans, fp);

	// Initialise inputs (generate sin wave in each subband
    srand(time(NULL));
    for(unsigned b = 0; b < nbeams; b++)
        for(unsigned sb = 0; sb < nsubs; sb++)
            for(unsigned i = 0; i < nsamp; i++)
            {
                input[b * nsubs * nsamp + sb * nsamp + i].x = sin(i * 0.1) + sin(i*0.5);// + rand() * 1e-9;
                input[b * nsubs * nsamp + sb * nsamp + i].y = 0;
            }

    // Write input file
    FILE *inFile = fopen("input.dat", "wb");
    fwrite(input, sizeof(hipComplex), nbeams * nsubs * nsamp, inFile);
    fclose(inFile);

    // Initialise CUDA stuff
    CudaSafeCall(hipSetDevice(1));
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    hipEvent_t event_start, event_stop;
    float timestamp;

    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop);

    // Allocate GPU memory
    CudaSafeCall(hipMalloc((void **) &d_input, nsamp * nsubs * nbeams * sizeof(hipComplex)));
	CudaSafeCall(hipMalloc((void **) &d_output, nsamp * nsubs * nbeams * sizeof(float)));
    CudaSafeCall(hipMalloc((void **) &d_weights, NTAPS * nchans * sizeof(float)));

    time_t start = time(NULL);

    // Copy input to GPU
    hipEventRecord(event_start, 0);
    CudaSafeCall(hipMemcpy(d_input, input, nsamp * nbeams * nsubs * sizeof(hipComplex), hipMemcpyHostToDevice));    
	CudaSafeCall(hipMemcpy(d_weights, weights, nchans * NTAPS * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied to GPU in: %lf \n", timestamp); 

    // Phase 1, perform FIR (apply window)
    hipEventRecord(event_start, 0);
    unsigned num_threads = 128;
    dim3 grid(nsubs, nbeams);
    ppf_fir<<<grid, num_threads, NTAPS * num_threads * sizeof(float)>>>(d_input, d_weights, nsamp / nchans, nsubs, nbeams, nchans);
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Performed FIR in: %lf [%.2f Gflops] \n", timestamp, (4 * NTAPS * nsubs * (nbeams * nsamp * 1.0e-9)) * (1.0 / (timestamp * 0.001)));

	// Phase 2, perform FFT
	hipfftHandle plan;
    hipfftPlan1d(&plan, nchans, HIPFFT_C2C, nsubs * nsamp / nsubs); // Plan only created once
   
	hipEventRecord(event_start, 0);
	for (unsigned i = 0; i < nbeams; i++)
        hipfftExecC2C(plan, d_input + i * nsubs * nsamp, d_input + i * nsubs * nsamp, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
	printf("Performed FFT in: %lf\n", timestamp);

    hipfftDestroy(plan);

	// Phase 3, fix channelisation order
//	hipEventRecord(event_start, 0);
//	dim3 fixDim(nsamp / nchans, nsubs, nbeams);  
//    fix_channelisation<<< fixDim, nchans >>> 
//                      (d_input, d_output, nsamp, nsubs, nbeams, nsubs, 0);
//	hipEventRecord(event_stop, 0);
//    hipEventSynchronize(event_stop);
//    hipEventElapsedTime(&timestamp, event_start, event_stop);
//	printf("Reordered channels in: %lf [%.2f Gflops]\n", timestamp,  (10 * nsubs * (nbeams * nsamp * 1.0e-9)) * (1.0 / (timestamp * 0.001)));

    // Copy output from GPU
    hipEventRecord(event_start, 0);
    CudaSafeCall(hipMemcpy(input, d_input, nsamp * nbeams * nsubs * sizeof(hipComplex), hipMemcpyDeviceToHost));    
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied to GPU in: %lf \n", timestamp);

    // Write test output file
    FILE *outFile = fopen("output.dat", "wb");
    fwrite(input, sizeof(hipComplex), nbeams * nsubs * nsamp, outFile);
    fclose(outFile);

    Exit();
}

