#include "hip/hip_runtime.h"
#include <cutil.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>

// ---------------------- Intensities Calculation Loop  ------------------------------

// One thread block per original subband (nsamp = nsamp * chansPerSubband)
__global__ __device__ void rfi_clipping(float *input, float *means, int nsamp, int nsubs)
{
    __shared__ float2 tempSums[1024];
    float mean, stddev;   // Store as registers to avoid bank conflicts in shared memory

    // Initial setup
    tempSums[threadIdx.x].x = 0;
    tempSums[threadIdx.x].y = 0;

    for(unsigned s =  threadIdx.x; 
                 s <  nsamp; 
                 s += blockDim.x)
    {
        // Calculate partial sums
        float intensity = input[blockIdx.y * nsamp + s];
        tempSums[threadIdx.x].x += intensity;
        tempSums[threadIdx.x].y += intensity * intensity;
    }

    // synchronise threads
    __syncthreads();

    // TODO: use reduction to optimise this part
    if (threadIdx.x == 0) {

        float val1, val2;
        for(unsigned i = 0; i < blockDim.x; i++) {
            val1 += tempSums[i].x;
            val2 += tempSums[i].y;
        }

        // Calculate mean and stddev
        mean   = val1 / nsamp;
        stddev = sqrtf((val2 - nsamp * mean * mean) / nsamp);
        means[blockIdx.y] = mean;

        // Store mean and stddev in tempSums
        tempSums[0].x = mean;
        tempSums[0].y = stddev;
    }

    // Synchronise threads
    __syncthreads();  
    mean = tempSums[0].x;
    stddev = tempSums[0].y;

    // Clip RFI within the subbands
    for(unsigned s =  threadIdx.x; 
                 s <  nsamp;    
                 s += blockDim.x)
    {
        float val = input[blockIdx.y * nsamp + s];
        float tempval = fabs(val - mean);
        if (tempval >= stddev * 4 || tempval <= stddev / 4)
            val = mean;

        __syncthreads();
        input[blockIdx.y * nsamp + s] = val;
    }
}

// ------------------------------------------------------------------------------------

int nsamp = 32, nsubs = 8;
int blocksize = 512;

// Process command-line parameters
void process_arguments(int argc, char *argv[])
{
    int i = 1;
    
    while((fopen(argv[i], "r")) != NULL)
        i++;

    while(i < argc) {
       if (!strcmp(argv[i], "-nsamp"))
           nsamp = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsubs"))
           nsubs = atoi(argv[++i]);
      else if (!strcmp(argv[i], "-blocksize"))
           blocksize = atoi(argv[++i]);
       i++;
    }
}

// -------------------------- Main Program -----------------------------------

int main(int argc, char *argv[]) 
{
    // Initialise stuff
    process_arguments(argc, argv);
    hipEvent_t event_start, event_stop;

    // Initialise CUDA stuff
    hipSetDevice(1);
    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop); 
    float timestamp;
    
    float *input, *d_input, *means, *d_means;
    
    printf("nsamp: %d, nsubs: %d\n", nsamp, nsubs);

    // Initialise data 
    input  = (float *) malloc(nsubs * nsamp * sizeof(float));
    means  = (float *) malloc(nsubs * sizeof(float));

    for(unsigned i = 0; i < nsubs; i++)
        for(unsigned j = 0; j < nsamp; j++)
            input[i * nsamp + j] = j;   

    // Allocate and transfer data to GPU (nsamp)
    hipMalloc((void **) &d_input,  sizeof(float) * nsubs * nsamp);
    hipMalloc((void **) &d_means,  sizeof(float) * nsubs);

    hipMemset(d_input, 0,  sizeof(float) * nsubs * nsamp);
    
    // Copy data to GPU
    hipEventRecord(event_start, 0);
    hipMemcpy(d_input, input, sizeof(float) * nsubs * nsamp, hipMemcpyHostToDevice);
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied input to GPU in: %lfms\n", timestamp);

    // Apply inter-subband clipping
    hipEventRecord(event_start, 0);
    rfi_clipping<<<dim3(1, nsubs), blocksize, blocksize*4 >>>
                          (d_input, d_means, nsamp, nsubs);
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Processed Intensities in: %lfms\n", timestamp);

    // Copy means to host memory
    hipMemcpy(means, d_means, sizeof(float) * nsubs, hipMemcpyDeviceToHost);
    for(unsigned i = 0; i < nsubs; i++) {
        if (means[i] != i)
            printf("Error: %d = %f\n", i, means[i]);
    }

    // Calculate mean of means
    double meanOfMeans;
    for(unsigned i = 0; i < nsubs; i++)
        meanOfMeans += means[i];
    meanOfMeans /= (nsubs * 1.0);
    printf("Mean of means: %lf\n", meanOfMeans);

    // Apply subband excision
    hipEventRecord(event_start, 0);

    for(unsigned i = 0; i < nsubs; i++)
        if (means[i] > 2 * meanOfMeans)
            hipMemset(d_input + i * nsamp, 0, nsamp * sizeof(float));

    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Subband RFI excision in: %lfms\n", timestamp);

    // Get result
    hipMemcpy(input, d_input, sizeof(float) * nsubs * nsamp, hipMemcpyDeviceToHost);
//    for(unsigned i = 0; i < nsubs; i++)
//        for(unsigned j=0; j < nsamp; j++)
//            printf("%d\t%d\t:  %f\n", i, j, input[i*nsamp+j]);
   
   // Clean up
   hipFree(d_input);
}
