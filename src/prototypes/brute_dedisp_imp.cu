#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "unistd.h"
#include "time.h"
#include "string.h"
#include <cutil_inline.h>

// Stores output value computed in inner loop for each thread
// __shared__ float shared[512];

// Stores temporary shift values
__constant__ float dm_shifts[4096];

// ---------------------- Optimised Dedispersion Loop  ------------------------------
__global__ void dedisperse_loop(float *outbuff, float *buff, int nsamp, int nchans, float tsamp,
                                float startdm, float dmstep, int maxshift)
{
    extern __shared__ float shared[];

    int c, s = threadIdx.x + blockIdx.x * blockDim.x;
    float shift_temp = (startdm + blockIdx.y * dmstep) / tsamp;
    
    for (s = threadIdx.x + blockIdx.x * blockDim.x; 
         s < nsamp; 
         s += blockDim.x * gridDim.x) {

        shared[threadIdx.x] = 0;
     
        for(c = 0; c < nchans; c++) {
            int shift = c * (nsamp + maxshift) + floor(dm_shifts[c] * shift_temp);
            shared[threadIdx.x] += buff[shift + s ];
        }

        outbuff[blockIdx.y * nsamp + s] = shared[threadIdx.x];
    }
}

// ----- Dodson's kernels  ----
__global__ void dispSearch_kernel(float* g_disp, float* g_data,
					float f0, float df, int fN,
					float dt, int tN,
					float dmin, float dmul)
{
	// get thread ids
	int i,j;
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int id_max = gridDim.x*blockDim.x;
	// get dispersion measure index
	int d = blockIdx.y;
	// get corresponding dispersion measure and multiply by the constant
	float kdm = 4.15e15*dmin*powf(dmul,d);
	if (dmul<0) kdm = 4.15e15*(dmin-d*dmul);
	// get max frequency
	float fM = f0+df*(fN-1);
	// divide all lags between the threads
	for (i=id; i<tN; i=i+id_max)
	{
		// add up along the candidate dm
		float sum = 0.0;
		for(j=0;j<fN;j++) {
			// get physical frequency value
			float f = f0+df*j;
			// get physical time value
			float t = kdm*(1/(f*f)-1/(fM*fM));
			// get array time offset
			int tloc = i + floorf(t/dt);
			if ((0<=tloc)&&(tloc<tN))
			{
				// and add it up
				sum += g_data[j*tN+tloc];
			}
		}
		// write sum to output
		g_disp[d*tN+i] = sum;
	}

}

__global__ void dispSearch_block_kernel(float* g_disp, float* g_data, 
					int sS, float s_0,
					float step, int sN, 
	   				int tN, int dN, int d_idx)
{
  int j,N;float sum,i;

  int id = blockDim.x*blockIdx.x + threadIdx.x;

  sum=0;
   {
    for (i=0;i<sN;i++) 
      { N=floorf(s_0-step*i+0.5);
	N=(id+N);//%tN;
	//N+=(i+sS)*tN;
	if ((N<tN)&&(N>=0)) {
	  sum += g_data[(int) (N+(i+sS)*tN)];
	}
     } }

   g_disp[id+tN*(d_idx)]=floorf(s_0-step*sN+0.5);;//s_0+sN*step;

}

// -------------------------- Main Program -----------------------------------


float fch1 = 156, foff = -0.005859375, tsamp = 0.000165, dmstep = 0.02, startdm = 0;
int nchans = 1024, nsamp = 1024, tdms = 1024;
int gridsize = 128, blocksize = 128;

// Process command-line parameters
void process_arguments(int argc, char *argv[])
{
    int i = 1;
    
    while((fopen(argv[i], "r")) != NULL)
        i++;

    while(i < argc) {
       if (!strcmp(argv[i], "-nchans"))
           nchans = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-nsamp"))
           nsamp = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-dmstep"))
           dmstep = atof(argv[++i]);
       else if (!strcmp(argv[i], "-startdm"))
           startdm = atof(argv[++i]);
       else if (!strcmp(argv[i], "-tdms"))
           tdms = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-gridsize"))
           gridsize = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-blocksize"))
           blocksize = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-tsamp"))
           blocksize = atoi(argv[++i]);
       else if (!strcmp(argv[i], "-foff"))
           foff = -atof(argv[++i]);
       i++;
    }
}

// Fill buffer with data (blocking call)
void generate_data(float* buffer, int nsamp, int nchans)
{
    for(int i = 0; i < nsamp * nchans; i++)
        buffer[i] = 0.1;
}

// DM delay calculation
float dmdelay(float f1, float f2)
{
  return(4148.741601 * ((1.0 / f1 / f1) - (1.0 / f2 / f2)));
}

int main(int argc, char *argv[])
{
   float *input, *output, *d_input, *d_output;
   int maxshift, i, j;

   process_arguments(argc, argv);

    // Calculate temporary DM-shifts
    float *dmshifts = (float *) malloc(nchans * sizeof(float));
    for (unsigned i = 0; i < nchans; i++)
          dmshifts[i] = dmdelay(fch1 + (foff * i), fch1);

    // Calculate maxshift
    maxshift = ceil(dmshifts[nchans - 1] * (startdm + dmstep * tdms) / tsamp);

    // Allocate and initialise arrays
    input = (float *) malloc( (nsamp + maxshift) * nchans * sizeof(float));
    output = (float *) malloc( nsamp * tdms * sizeof(float));
    for(i = 0; i < nchans; i++)
        for(j = 0; j < nsamp + maxshift; j++) {
            input[i * (nsamp + maxshift) + j] = i;
         }

    // Initialise CUDA stuff
    cutilSafeCall( hipSetDevice(1));
    hipEvent_t event_start, event_stop;
    float timestamp, kernelTime;

    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop);

   printf("nsamp: %d, nchans: %d, tsamp: %f, startdm: %f, dmstep: %f, tdms: %d, fch1: %f, foff: %f, maxshift: %d\n",
           nsamp, nchans, tsamp, startdm, dmstep, tdms, fch1, foff, maxshift);

    // Allocate CUDA memory and copy dmshifts
    cutilSafeCall( hipMalloc((void **) &d_input, (nsamp + maxshift) * nchans * sizeof(float)));
    cutilSafeCall( hipMalloc((void **) &d_output, nsamp * tdms * sizeof(float)));
    cutilSafeCall( hipMemset(d_output, 0, nsamp * tdms * sizeof(float)));
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(dm_shifts), dmshifts, nchans * sizeof(int)) );

    time_t start = time(NULL);

    // Copy input to GPU
    hipEventRecord(event_start, 0);
    cutilSafeCall( hipMemcpy(d_input, input, (nsamp + maxshift) * nchans * sizeof(float), hipMemcpyHostToDevice) );    
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied to GPU in: %lf\n", timestamp);

//    dim3 gridDim(nsamp / blocksize, tdms);  
//    hipEventRecord(event_start, 0);
//    dedisperse_loop<<<gridDim, blocksize, 512>>>(d_output, d_input, nsamp, nchans, tsamp, startdm, dmstep, maxshift);
//    hipEventRecord(event_stop, 0);
//    hipEventSynchronize(event_stop);
//    hipEventElapsedTime(&timestamp, event_start, event_stop);
//    printf("Processed in: %lf\n", timestamp);
//    kernelTime = timestamp;
//     printf("Performance: %lf Gflops\n", (nchans * tdms) * (nsamp * 1.0 / kernelTime / 1.0e6));

    hipEventRecord(event_start, 0);
	dim3 block(128,1,1);
	dim3 grid(30,tdms,1);
    dispSearch_kernel<<<grid, block>>>(d_output, d_input, fch1, foff, nchans, tsamp, nsamp, startdm, dmstep); 
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Processed in: %lf\n", timestamp);
    kernelTime = timestamp;
    printf("Performance: %lf Gflops\n", (nchans * tdms) * ((nsamp - maxshift) * 1.0 / kernelTime / 1.0e6));

//// * @param g_disp Output DM-Lag space
//// * @param g_data Input data (assumed lag-contiguous, may need cornerTurn first)
//// * @param f0 frequency of the lowest channel
//// * @param df bandwidth per channel
//// * @param fN total number of channels
//// * @param dt time per sample
//// * @param tN total number of samples per channel
//// * @param dmin lowest dispersion measure
//// * @param dN number of dispersion measures (currently limited by grid dim)
//// * @param dmul multiplication factor for dispersion measures



    // Copy output from GPU
    hipEventRecord(event_start, 0);
    cutilSafeCall( hipMemcpy(output, d_output, nsamp * tdms * sizeof(float), hipMemcpyDeviceToHost) );    
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&timestamp, event_start, event_stop);
    printf("Copied from GPU in: %lf\n", timestamp);

    int val = 0;
    for(i = 0; i < nchans; i++) val += i;

//    for(i = 0; i < tdms; i++)
//        for(j = 0; j < nsamp; j++)
//            if (output[i * nsamp + j] != val)
//                printf("Error: dm: %d nsamp: %d value:%f \n", i, j, output[i*nsamp+j]);

    printf("Total time: %d\n", (int) (time(NULL) - start));
}

