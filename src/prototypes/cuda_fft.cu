#include <cutil.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>

#define NX 512 * 1024
#define NY 4 * 1024
#define BATCH 1

int main() {

    // Initialise stuff
    hipEvent_t event_start, event_stop;
    struct timeval start, stop;
    struct timezone tzp;
    int i;

    // Initialise CUDA stuff
    hipSetDevice(1);
    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop); 

    // ==================== 1D FFT ==============================

    hipfftHandle plan;
    hipfftComplex *d_data, *data;

    // Initialise data 
    data = (hipfftComplex *) malloc(NX * BATCH * sizeof(hipfftComplex));
    for(i = 0; i < NX * BATCH; i++) {
        data[i].x = 1.0f;
        data[i].y = 1.0f;
    }

   // Allocate and transfer data to GPU
   hipMalloc((void **) &d_data, sizeof(hipfftComplex) * NX * BATCH);
   hipMemcpy(d_data, data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);

   // Create plan
   hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);

   // Execute FFT
   gettimeofday(&start, &tzp);
   hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
   hipDeviceSynchronize();
   gettimeofday(&stop, &tzp);
   if (start.tv_usec > stop.tv_usec) {
       stop.tv_usec += 1000000;
       stop.tv_sec--;
   }
   printf("Processed 1D FFT in %0.2fms [%d, %d]\n", (stop.tv_usec - start.tv_usec) / 1000.0f, NX, BATCH);
 
   // Get result
   hipMemcpy(data, d_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

   // Clean up
   hipfftDestroy(plan);
   hipFree(d_data);

 // ==================== 2D FFT ==============================

    // Initialise data 
    realloc(data, NX * BATCH * sizeof(hipfftComplex));
    for(i = 0; i < NX * BATCH; i++) {
        data[i].x = 1.0f;
        data[i].y = 1.0f;
    }

   // Allocate and transfer data to GPU
   hipMalloc((void **) &d_data, sizeof(hipfftComplex) * NX * BATCH);
   hipMemcpy(d_data, data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);

   // Create plan
   hipfftPlan2d(&plan, NY, NY, HIPFFT_C2C);

   // Execute FFT
   gettimeofday(&start, &tzp);
   hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
   hipDeviceSynchronize();
   gettimeofday(&stop, &tzp);
   if (start.tv_usec > stop.tv_usec) {
       stop.tv_usec += 1000000;
       stop.tv_sec--;
   }
   printf("Processed 2D FFT in %0.2fms [%d, %d]\n", (stop.tv_usec - start.tv_usec) / 1000.0f, NY, NY);
 
   // Get result
   hipMemcpy(data, d_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

   // Clean up
   hipfftDestroy(plan);
   hipFree(d_data);

}
